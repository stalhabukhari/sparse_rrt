#include "hip/hip_runtime.h"
#include "trajectory_optimizers/cem_cuda_acrobot.hpp"
#include "systems/enhanced_system.hpp"

//#define PROFILE

#include <chrono>

// #define DEBUG

// define system parameters
#include <cmath>
#define LENGTH 20.0
#define m 1.0

#define lc  .5
#define lc2  .25
#define l2  1
#define I1  0.2
#define I2  1.0
#define l  1.0
#define g  9.8



#define STATE_THETA_1 0
#define STATE_THETA_2 1
#define STATE_V_1 2
#define STATE_V_2 3
#define CONTROL_T 0

#define MIN_V_1 -6
#define MAX_V_1 6
#define MIN_V_2 -6
#define MAX_V_2 6
#define MIN_TORQUE -4
#define MAX_TORQUE 4


#define DT  2e-2

#define PI  3.141592654f

#define DIM_STATE 4
#define DIM_CONTROL 1
#define NOBS 4
#define OBS_PENALTY 1000.0





namespace trajectory_optimizers_acrobot{

    __global__ void initCurand(hiprandState* state, unsigned long seed) {
        int idx = threadIdx.x + blockIdx.x * blockDim.x;
        hiprand_init(seed, idx, 0, &state[idx]);
    }


    __global__ 
    void set_statistics(double* d_mean_time, const double mean_time, double* d_mean_control, const double* mean_control, 
        double* d_std_control, const double* std_control, double* d_std_time, const double std_time, int NT){
        unsigned int np = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int nt = blockIdx.z * blockDim.z + threadIdx.z;
        unsigned int id = np * NT + nt;
        //unsigned int id = blockIdx.x*blockDim.x + threadIdx.x;// 0~NT * NP
        //printf("inside set_statistics. id: (%d, %d)\n", np, nt);

        d_mean_time[id] = mean_time;
        d_mean_control[id] = mean_control[0];
        d_std_control[id] = std_control[0];
        d_std_time[id] = std_time;
        // printf("inside set_statistics. d_mean_time: %f\n", d_mean_time[id]);
        // printf("inside set_statistics. d_mean_control: %f\n", d_mean_control[id]);
        // printf("inside set_statistics. d_std_control: %f\n", d_std_control[id]);
        // printf("inside set_statistics. d_std_time: %f\n", d_std_time[id]);

        
    }

    __global__
    void set_start_state(double* temp_state, double* start, const int NS){
        unsigned int np = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int ns = blockIdx.y * blockDim.y + threadIdx.y;
        unsigned int id = np * NS + ns;
    
        temp_state[STATE_THETA_1 + id*DIM_STATE] = start[STATE_THETA_1 + np* DIM_STATE];
        temp_state[STATE_THETA_2 + id*DIM_STATE] = start[STATE_THETA_2 + np * DIM_STATE];
        temp_state[STATE_V_1 + id*DIM_STATE] = start[STATE_V_1 + np * DIM_STATE];
        temp_state[STATE_V_2 + id*DIM_STATE] = start[STATE_V_2 + np * DIM_STATE]; 
        //printf("%d: %f, %f, %f, %f\n", id, temp_state[id * DIM_STATE + 0], temp_state[id * DIM_STATE + 1], temp_state[id * DIM_STATE + 2], temp_state[id * DIM_STATE + 3]);

    }

    __global__ 
    void sampling(double* control, double* time, double* mean_control, double* mean_time, double* std_control, double* std_time, double MAX_T, const int NP, const int NS, const int NT, bool* active_mask,
        hiprandState* state){
        unsigned int np = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int ns = blockIdx.y * blockDim.y + threadIdx.y;
        unsigned int nt = blockIdx.z * blockDim.z + threadIdx.z;
        unsigned int id = np * NS * NT + ns * NT + nt;

        //printf("%d, %d, %d\n",np, ns, nt);
        //printf("inside sampling. id: (%d, %d, %d)\n", np, ns, nt);

        active_mask[np * NS + ns] = true;
        //printf("inside sampling. hiprand_normal control: %f\n", hiprand_normal(&state[id]) + mean_control[np * NT + nt]);
        //printf("inside sampling. std_control: %f\n", std_control[np * NT + nt]);

        double c = std_control[np * NT + nt] * hiprand_normal(&state[id]) + mean_control[np * NT + nt];
        //printf("mean control:%f\n", mean_control[np * NT + nt]);

        if (c > MAX_TORQUE) {
            c = MAX_TORQUE;
        }
        else if (c < MIN_TORQUE) {
            c = MIN_TORQUE;
        }
        control[np * NS * NT + ns * NT + nt] = c;

        //printf("inside sampling. hiprand_normal time: %f\n", std_time[np * NT + nt] * hiprand_normal(&state[id]) + mean_time[np * NT + nt]);
        //printf("inside sampling. std_time: %f\n", std_time[np * NT + nt]);
        //printf("inside sampling. mean_time: %f\n", mean_time[np * NT + nt]);
        double t = std_time[np * NT + nt] * hiprand_normal(&state[id]) + mean_time[np * NT + nt];
        //if(t < DT){  // original working one (on cartpole)
        if (t < 0.){
            t = 0;
        } else if (t > MAX_T) {
            t = MAX_T;
        }
        time[np * NS * NT + ns * NT + nt] = t;      
        //printf("c:%f, t:%f\n", c, t);

    }

    __device__
    bool lineLine(double x1, double y1, double x2, double y2, double x3, double y3, double x4, double y4)
    // compute whether two lines intersect with each other
    {
        // ref: http://www.jeffreythompson.org/collision-detection/line-rect.php
        // calculate the direction of the lines
        double uA = ((x4-x3)*(y1-y3) - (y4-y3)*(x1-x3)) / ((y4-y3)*(x2-x1) - (x4-x3)*(y2-y1));
        double uB = ((x2-x1)*(y1-y3) - (y2-y1)*(x1-x3)) / ((y4-y3)*(x2-x1) - (x4-x3)*(y2-y1));
    
        // if uA and uB are between 0-1, lines are colliding
        if (uA >= 0 && uA <= 1 && uB >= 0 && uB <= 1)
        {
            // intersect
            return true;
        }
        // not intersect
        return false;
    }

    __device__
    bool valid_state(double* temp_state, double* obs_list)
    {
        // check the pole with the rectangle to see if in collision
        // calculate the pole state
        // check if the position is within bound
        double pole_x0 = 0.;
        double pole_y0 = 0.;
        double pole_x1 = (LENGTH) * cos(temp_state[STATE_THETA_1] - M_PI / 2);
        double pole_y1 = (LENGTH) * sin(temp_state[STATE_THETA_1] - M_PI / 2);
        double pole_x2 = pole_x1 + (LENGTH) * cos(temp_state[STATE_THETA_1] + temp_state[STATE_THETA_2] - M_PI / 2);
        double pole_y2 = pole_y1 + (LENGTH) * sin(temp_state[STATE_THETA_1] + temp_state[STATE_THETA_2] - M_PI / 2);

        //std::cout << "state:" << temp_state[0] << "\n";
        //std::cout << "pole point 1: " << "(" << pole_x1 << ", " << pole_y1 << ")\n";
        //std::cout << "pole point 2: " << "(" << pole_x2 << ", " << pole_y2 << ")\n";
        for(unsigned int i = 0; i < NOBS; i++)
        {
            // check if any obstacle has intersection with pole
            //std::cout << "obstacle " << i << "\n";
            //std::cout << "points: \n";
            for (unsigned int j = 0; j < 8; j+=2)
            {

                //std::cout << j << "-th point: " << "(" << obs_list[i][j] << ", " << obs_list[i][j+1] << ")\n";
            }
            for (unsigned int j = 0; j < 8; j+=2)
            {
                // check each line of the obstacle
                double x1 = obs_list[i * 8 + j];
                double y1 = obs_list[i * 8 + j + 1];
                double x2 = obs_list[i * 8 + (j+2) % 8];
                double y2 = obs_list[i * 8 +(j+3) % 8];
                if (lineLine(pole_x0, pole_y0, pole_x1, pole_y1, x1, y1, x2, y2))
                {
                    // intersect
                    return false;
                }
                if (lineLine(pole_x1, pole_y1, pole_x2, pole_y2, x1, y1, x2, y2))
                {
                    // intersect
                    return false;
                }
            }
        }
        return true;
    }

    __global__
    void propagate(double* temp_state, double* control, double* time, double* deriv, 
        const int t_step, const int NS, const int NT, bool* active_mask, double* obs_list){
            unsigned int np = blockIdx.x * blockDim.x + threadIdx.x;
            unsigned int ns = blockIdx.y * blockDim.y + threadIdx.y;
            unsigned int id = np * NS + ns;
            //printf("%d, %d, %d, %d\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y);
            //unsigned int id = blockIdx.x*blockDim.x + threadIdx.x;
                //printf("%d\n", id);

            double t = time[np * NS * NT + ns * NT + t_step];
            if (t < 0){
                t = 0;
            }
            int num_step = t / DT;
            double _a = control[np * NS * NT + ns * NT + t_step];
                
            for(unsigned int i = 0; i < num_step; i++){
                if(!active_mask[id]){
                    break;
                }
                // update derivs
                double theta2 = temp_state[STATE_THETA_2 + id*DIM_STATE];
                double theta1 = temp_state[STATE_THETA_1 + id*DIM_STATE] - M_PI / 2;
                double theta1dot = temp_state[STATE_V_1 + id*DIM_STATE];
                double theta2dot = temp_state[STATE_V_2 + id*DIM_STATE];
                double _tau = _a;
            
                //extra term m*lc2
                double d11 = m * lc2 + m * (l2 + lc2 + 2 * l * lc * cos(theta2)) + I1 + I2;
            
                double d22 = m * lc2 + I2;
                double d12 = m * (lc2 + l * lc * cos(theta2)) + I2;
                double d21 = d12;
            
                //extra theta1dot
                double c1 = -m * l * lc * theta2dot * theta2dot * sin(theta2) - (2 * m * l * lc * theta1dot * theta2dot * sin(theta2));
                double c2 = m * l * lc * theta1dot * theta1dot * sin(theta2);
                double g1 = (m * lc + m * l) * g * cos(theta1) + (m * lc * g * cos(theta1 + theta2));
                double g2 = m * lc * g * cos(theta1 + theta2);
            
                deriv[STATE_THETA_1 + id*DIM_STATE] = theta1dot;
                deriv[STATE_THETA_2 + id*DIM_STATE] = theta2dot;
            
                double u2 = _tau - 1 * .1 * theta2dot;
                double u1 = -1 * .1 * theta1dot;
                double theta1dot_dot = (d22 * (u1 - c1 - g1) - d12 * (u2 - c2 - g2)) / (d11 * d22 - d12 * d21);
                double theta2dot_dot = (d11 * (u2 - c2 - g2) - d21 * (u1 - c1 - g1)) / (d11 * d22 - d12 * d21);
            
                deriv[STATE_V_1 + id*DIM_STATE] = theta1dot_dot;
                deriv[STATE_V_2 + id*DIM_STATE] = theta2dot_dot;

                // update states
                temp_state[STATE_THETA_1 + id*DIM_STATE] += DT * deriv[STATE_THETA_1 + id*DIM_STATE];
                temp_state[STATE_THETA_2 + id*DIM_STATE] += DT * deriv[STATE_THETA_2 + id*DIM_STATE];
                temp_state[STATE_V_1 + id*DIM_STATE] += DT * deriv[STATE_V_1 + id*DIM_STATE];
                temp_state[STATE_V_2 + id*DIM_STATE] += DT * deriv[STATE_V_2 + id*DIM_STATE];
                // enforce bounds

                if(temp_state[0 + id*DIM_STATE]<-M_PI)
                        temp_state[0 + id*DIM_STATE]+=2*M_PI;
                else if(temp_state[0 + id*DIM_STATE]>M_PI)
                        temp_state[0 + id*DIM_STATE]-=2*M_PI;
                if(temp_state[1 + id*DIM_STATE]<-M_PI)
                        temp_state[1 + id*DIM_STATE]+=2*M_PI;
                else if(temp_state[1 + id*DIM_STATE]>M_PI)
                        temp_state[1 + id*DIM_STATE]-=2*M_PI;
                if(temp_state[2 + id*DIM_STATE]<MIN_V_1)
                        temp_state[2 + id*DIM_STATE]=MIN_V_1;
                else if(temp_state[2 + id*DIM_STATE]>MAX_V_1)
                        temp_state[2 + id*DIM_STATE]=MAX_V_1;
                if(temp_state[3 + id*DIM_STATE]<MIN_V_2)
                        temp_state[3 + id*DIM_STATE]=MIN_V_2;
                else if(temp_state[3 + id*DIM_STATE]>MAX_V_2)
                        temp_state[3 + id*DIM_STATE]=MAX_V_2;
                // validate_states
                bool valid = valid_state(&temp_state[id*DIM_STATE], obs_list);
                active_mask[id] = active_mask[id] && valid;
            }        
            //printf("%d, %d: %f, %f, %f, %f\n", ns, np, temp_state[id * DIM_STATE + 0], temp_state[id * DIM_STATE + 1], temp_state[id * DIM_STATE + 2], temp_state[id * DIM_STATE + 3]);

    }

    __global__
    void get_loss(double* temp_state, double* loss, const int NS, double* goal_state, bool* active_mask){
        //printf("%d\n", id);
        unsigned int np = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int ns = blockIdx.y * blockDim.y + threadIdx.y;
        unsigned int id = np * NS + ns;

        double loss1= temp_state[id * DIM_STATE + STATE_THETA_1] - goal_state[np * DIM_STATE + STATE_THETA_1];
        if (loss1 < 0)
        {
            loss1 = -loss1;
        }
        if (loss1 > M_PI)
        {
            loss1 = 2*M_PI - loss1;
        }
        double loss2 = temp_state[id * DIM_STATE + STATE_THETA_2] - goal_state[np * DIM_STATE + STATE_THETA_2];
        if (loss2 < 0)
        {
            loss2 = -loss2;
        }
        if (loss2 > M_PI)
        {
            loss2 = 2*M_PI - loss2;
        }       
        loss[id] = sqrt(loss1*loss1 + loss2*loss2 \
            + 0.5 * (temp_state[id * DIM_STATE + STATE_V_1] - goal_state[np * DIM_STATE + STATE_V_1]) * (temp_state[id * DIM_STATE + STATE_V_1] - goal_state[np * DIM_STATE + STATE_V_1])\
            + 0.5 * (temp_state[id * DIM_STATE + STATE_V_2] - goal_state[np * DIM_STATE + STATE_V_2]) * (temp_state[id * DIM_STATE + STATE_V_2] - goal_state[np * DIM_STATE + STATE_V_2]));
        if (!active_mask[id]) {
            loss[id] += OBS_PENALTY;
        }
        //printf("loss[%d]: %f", id, loss[id]);

        // printf("%d, %d: %f, %f, %f, %f, loss: %f\n", 
        //     ns, np, 
        //     temp_state[id * DIM_STATE + 0], temp_state[id * DIM_STATE + 1], temp_state[id * DIM_STATE + 2], temp_state[id * DIM_STATE + 3],
        //     loss[id]);

    }

    
    __global__
    void get_approx_topk_loss(double* loss, const int NS, double* top_k_loss, int* top_k_loss_ind, const int N_ELITE){
        //printf("%d\n", id);
        /**
        * #TODO
        * this uses the id to map to one of the k buckets, and then only find the min loss for that index.
        * this is approximate as the result may not be the top k.
        * for consistency against previous method, our inputs are of the following dimension:
        *       top_k_loss: NPxNS
        *       top_k_loss_ind: NPxNS
        * Since we have NP problems, our bucket is of size: NP x N_ELITE
        *       grid size: (1, 1, 1)
        *       block size: (NP, 1, NE)
        */
        unsigned int np = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int ne = blockIdx.z * blockDim.z + threadIdx.z;

        // loop over NE to find best k
        double min_loss = 10*OBS_PENALTY;
        int min_loss_ind = -1;
        for (unsigned int ns_div_ne = 0; ns_div_ne < NS/N_ELITE; ns_div_ne++)
        {
            unsigned int ns = ns_div_ne * N_ELITE + ne;
            if (ns >= NS)
            {
                continue;
            }
            if (loss[np*NS + ns] < min_loss)
            {
                min_loss = loss[np*NS + ns];
                min_loss_ind = ns;
            }
        }
        // copy the min loss to the bucket
        top_k_loss[np*NS+ne] = min_loss;
        top_k_loss_ind[np*NS+ne] = min_loss_ind;
    }


    __global__
    void update_statistics(double* control, double* time, double* mean_control, double* mean_time, double* std_control, double* std_time,
        int* loss_ind, double* loss, int NP, int NS, int NT, int N_ELITE, double* best_ut){
        unsigned int np = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int nt = blockIdx.z * blockDim.z + threadIdx.z;
        //printf("inside update_statistics. id: (%d, %d)\n", np, nt);

        //unsigned int id = blockIdx.x*blockDim.x + threadIdx.x;
        double sum_control = 0., sum_time = 0., ss_control = 0., ss_time = 0.;
        for(int i = 0; i < N_ELITE; i++){
            //printf("inside update_statistics. N_ELITE: %d\n", N_ELITE);
            //printf("inside update_statistics. elite_i: %d\n", i);
            unsigned int id = np * NS * NT + loss_ind[np * NS + i] * NT + nt;
            //printf("inside update_statistics. loss: %f\n", loss[np * NS + i]);
            //printf("inside update_statistics. loss_ind: %d\n", loss_ind[np * NS + i]);
            //printf("inside update_statistics. id: %d\n", id);

            sum_control += control[id];
            ss_control += control[id] * control[id];
            sum_time += time[id];
            ss_time += time[id] * time[id];
        }
        // printf("%f,%f\n",ss_control, ss_time);
        unsigned int s_id = np * NT + nt;
        mean_control[s_id] = sum_control / N_ELITE;
        mean_time[s_id] = sum_time / N_ELITE;
        double std_control_square = ss_control / N_ELITE - mean_control[s_id] * mean_control[s_id];
        if (std_control_square < 1e-5)
        {
            std_control_square = 1e-5;
        }
        std_control[s_id] = sqrt(std_control_square);

        //printf("inside update_statistics. ss_time: %f\n", ss_time);
        //printf("inside update_statistics. ss_time/N_ELITE: %f\n", ss_time/N_ELITE);

        double std_time_square = ss_time / N_ELITE - mean_time[s_id] * mean_time[s_id];
        if (std_time_square < 1e-5)
        {
            std_time_square = 1e-5;
        }
        std_time[s_id] = sqrt(std_time_square);
        
        //printf("inside update_statistics. ss_time: %f\n", ss_time);
        //printf("inside update_statistics. ss_time/N_ELITE: %f\n", ss_time/N_ELITE);
        //printf("inside update_statistics. std_time_square: %f\n", ss_time / N_ELITE - mean_time[s_id] * mean_time[s_id]);
        //printf("inside update_statistics. mean_time: %f\n", mean_time[s_id]);
        //printf("inside update_statistics. mean_time_square: %f\n", mean_time[s_id] * mean_time[s_id]);

        //printf("inside update_statistics. std_time: %f\n", std_time[s_id]);


        best_ut[s_id] = control[np * NS * NT + loss_ind[np * NS] * NT + nt];
        best_ut[s_id + NP * NT] = time[np * NS * NT + loss_ind[np * NS] * NT + nt];
        //printf("inside update_statistics. best_u[s_id]: %f\n",  best_ut[s_id]);
        //printf("inside update_statistics. best_t[s_id]: %f\n",  best_ut[s_id + NP * NT]);

    }
    

    CEM_CUDA_acrobot::CEM_CUDA_acrobot(enhanced_system_t* model, unsigned int num_of_problems, unsigned int number_of_samples, unsigned int number_of_t,
        unsigned int number_of_elite,  double converge_r,
        std::vector<std::vector<double>>& _obs_list,
        double* control_means, double* control_stds, 
        double time_means, double time_stds, double max_duration,
        double integration_step, double* loss_weights, unsigned int max_iteration, bool verbose, double step_size)
        : trajectory_optimizers::CEM(model, number_of_samples, number_of_t,
            number_of_elite, converge_r, 
            control_means, control_stds, 
            time_means, time_stds, max_duration,
            integration_step, loss_weights, max_iteration, verbose, step_size)
    {
        /*
        * TODO:
        * for parent class, remove redundent members initialization and creation
        */



        system = model;
        this -> NP = num_of_problems;
        this -> NS = number_of_samples;
        this -> NT = number_of_t;
        this -> N_ELITE = number_of_elite;
        mu_u0 = new double[DIM_CONTROL]();
        std_u0 = new double[DIM_CONTROL]();
        for (unsigned i=0; i < DIM_CONTROL; i++)
        {
            mu_u0[i] = control_means[i];
            std_u0[i] = control_stds[i];
        }
        mu_t0 = time_means;
        std_t0 = time_stds;
        this -> max_duration = max_duration;
        s_dim = system -> get_state_dimension();
        c_dim = system -> get_control_dimension();
        dt = integration_step;

        // states for rolling
        this -> verbose = verbose;
        converge_radius = converge_r;
        
        // util variables for update statistics
        step_size = step_size;
        it_max = max_iteration;
        weight = new double[s_dim];
        for(unsigned int si = 0; si < s_dim; si++){
            weight[si] = loss_weights[si];
        }

        double width = 6;

        // for CUDA here
        //printf("setup...\n");
        // best_ut = (double*) malloc(2 * NP * NT /*time + control*/ * sizeof(double));   // 2 x NP x NT
        hipMalloc(&d_best_ut, NP * NT * 2 * sizeof(double)); 

        hipMalloc(&d_mu_u0, DIM_CONTROL * sizeof(double)); 
        hipMalloc(&d_std_u0, DIM_CONTROL * sizeof(double)); 
        hipMemcpy(d_mu_u0, mu_u0, DIM_CONTROL * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_std_u0, std_u0, DIM_CONTROL * sizeof(double), hipMemcpyHostToDevice);

        // temp state, derivative, control, time samples
            // temp_state = (double*) malloc(NS * DIM_STATE * sizeof(double));
        hipMalloc(&d_temp_state, NP * NS * DIM_STATE * sizeof(double)); 
        hipMalloc(&d_deriv, NP * NS * DIM_STATE * sizeof(double));
        hipMalloc(&d_control, NP * NS * NT * DIM_CONTROL * sizeof(double));
        hipMalloc(&d_time, NP * NS * NT * sizeof(double));
        // for sampling statistics
        hipMalloc(&d_mean_time, NP * NT * sizeof(double));
        hipMalloc(&d_mean_control, NP * NT* sizeof(double));
        hipMalloc(&d_std_control, NP * NT * sizeof(double));
        hipMalloc(&d_std_time, NP * NT * sizeof(double));
        // for cem
        hipMalloc(&d_loss, NP * NS * sizeof(double));

        hipMalloc(&d_top_k_loss, NP * NS * sizeof(double)); 

        hipMalloc(&d_loss_ind, NP * NS * sizeof(int));
        //loss_ind = (int*) malloc(NP * NS * sizeof(int));
        loss_ind = new int[NP*NS]();
        //memset(loss_ind, 0, NP * NS  * sizeof(int));
        
        loss = new double[NP*NS]();
        loss_pair.resize(NS, std::make_pair(0., 0));

        // obstacles
        hipMalloc(&d_obs_list, NOBS * 8 * sizeof(double));
        hipMalloc(&d_active_mask, NP * NS * sizeof(bool));

        
        
        obs_list = new double[NOBS*8]();
        for(unsigned i=0; i<_obs_list.size(); i++)
        {
            // each obstacle is represented by its middle point
            // calculate the four points representing the rectangle in the order
            // UL, UR, LR, LL
            // the obstacle points are concatenated for efficient calculation
            double x = _obs_list[i][0];
            double y = _obs_list[i][1];
            //std::cout << x <<","<< y << std::endl;
            obs_list[i*8 + 0] = x - width / 2;  obs_list[i*8 + 1] = y + width / 2;
            obs_list[i*8 + 2] = x + width / 2;  obs_list[i*8 + 3] = y + width / 2;
            obs_list[i*8 + 4] = x + width / 2;  obs_list[i*8 + 5] = y - width / 2;
            obs_list[i*8 + 6] = x - width / 2;  obs_list[i*8 + 7] = y - width / 2;

        }
        hipMemcpy(d_obs_list, obs_list, sizeof(double) * NOBS * 8, hipMemcpyHostToDevice);
        // for multiple start
        hipMalloc(&d_start_state, NP * DIM_STATE * sizeof(double));
        hipMalloc(&d_goal_state, NP * DIM_STATE * sizeof(double));

        // initiate hiprand
        hipMalloc((void**)&devState,NP * NS * NT * sizeof(hiprandState));
        initCurand << <(NP * NS * NT + 31) / 32, 32 >> > (devState, 42);
        
        //printf("done, execution:\n");

    }
    void CEM_CUDA_acrobot::solve(const double* start, const double* goal, double* best_u, double* best_t){
        // auto begin = std::chrono::system_clock::now();
        // start and goal should be NP * DIM_STATE
        //std::cout << "inside CEM_CUDA::solve" << std::endl;
         #ifdef PROFILE
        auto profile_start = std::chrono::high_resolution_clock::now();
        #endif
        hipMemcpy(d_start_state, start, NP * DIM_STATE * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_goal_state, goal, NP * DIM_STATE * sizeof(double), hipMemcpyHostToDevice);
        //thrust::device_ptr<double> time_ptr(d_time);
        //thrust::device_ptr<double> control_ptr(d_control);
        #ifdef PROFILE

        auto profile_stop = std::chrono::high_resolution_clock::now();
        std::chrono::duration<float> profile_duration = profile_stop - profile_start; 
        std::cout << "inside cem_cuda:solve. hipMemcpy start & goal takes " << profile_duration.count() << "s" << std::endl; 
        std::cout << "inside cem_cuda:solve. 1000 steps of hipMemcpy start & goal takes " << 1000*profile_duration.count() << "s" << std::endl; 
        #endif        
        dim3 grid(1, 1, 1);
        dim3 grid_s(1, NS, 1);

        dim3 block_pt(NP, 1, NT);
        dim3 block_p(NP, 1, 1);

        //thrust::device_ptr<double> loss_ptr(d_loss);
        //thrust::device_ptr<int> loss_ind_ptr(d_loss_ind);
        //init mean
        //printf("%f,%f,%f,%f\n", mu_t0, std_t0, mu_u0[0], std_u0[0]);
        #ifdef PROFILE

        profile_start = std::chrono::high_resolution_clock::now();
        #endif

        set_statistics<<<grid, block_pt>>>(d_mean_time, mu_t0, d_mean_control, d_mu_u0, d_std_control, d_std_u0, d_std_time, std_t0, NT);

        #ifdef PROFILE
        profile_stop = std::chrono::high_resolution_clock::now();
        profile_duration = profile_stop - profile_start; 
        std::cout << "inside cem_cuda:solve. set_statistics takes " << profile_duration.count() << "s" << std::endl; 
        std::cout << "inside cem_cuda:solve. 1000 steps of set_statistics takes " << 1000*profile_duration.count() << "s" << std::endl; 
        #endif
        // double min_loss = 1e5;
        // double tmp_min_loss = 2e5;
        // auto init_end = std::chrono::system_clock::now();
        //std::cout<< "start" <<std::endl;
       

        for(unsigned int it = 0; it < it_max; it ++){
            //std::cout << "iteration: " << it << std::endl; 
            set_start_state<<<grid_s, block_p>>>(d_temp_state, d_start_state, NS);
            #ifdef PROFILE

            profile_start = std::chrono::high_resolution_clock::now();
            #endif
            sampling << <grid_s, block_pt >> > (d_control, d_time, d_mean_control, d_mean_time, d_std_control, d_std_time, max_duration, NP, NS, NT, d_active_mask, devState);
            #ifdef PROFILE
            profile_stop = std::chrono::high_resolution_clock::now();
            profile_duration = profile_stop - profile_start; 
            std::cout << "inside cem_cuda:solve. sampling takes " << profile_duration.count() << "s" << std::endl; 
            std::cout << "inside cem_cuda:solve. 1000 steps of sampling takes " << 1000*profile_duration.count() << "s" << std::endl; 
            #endif
            //std::cout<< "start of propagation..." <<std::endl;
            #ifdef PROFILE

            profile_start = std::chrono::high_resolution_clock::now();
            #endif
            for(unsigned int t_step = 0; t_step < NT; t_step++){
                propagate<<<grid_s, block_p >>>(d_temp_state, d_control, d_time, d_deriv, t_step, NS, NT, d_active_mask, d_obs_list);
            }
            #ifdef PROFILE

            profile_stop = std::chrono::high_resolution_clock::now();
            profile_duration = profile_stop - profile_start; 
            //std::cout << "inside cem_cuda:solve. propagate takes " << profile_duration.count() << "s" << std::endl; 
            //std::cout << "inside cem_cuda:solve. 1000 steps of propagate takes " << 1000*profile_duration.count() << "s" << std::endl; 
            #endif


            //std::cout<< "end of propagation." <<std::endl;
            #ifdef PROFILE

            profile_start = std::chrono::high_resolution_clock::now();
            #endif
            get_loss<<< grid_s, block_p >>>(d_temp_state, d_loss, NS, d_goal_state, d_active_mask);
            
            #ifdef PROFILE
            profile_stop = std::chrono::high_resolution_clock::now();
            profile_duration = profile_stop - profile_start; 
            std::cout << "inside cem_cuda:solve. get_loss takes " << profile_duration.count() << "s" << std::endl; 
            std::cout << "inside cem_cuda:solve. 1000 steps of get_loss takes " << 1000*profile_duration.count() << "s" << std::endl; 
            #endif

            /*
            //**  below method converts GPU to CPU, sorts in CPU, and then converts back
            // copy gpu loss to cpu

            #ifdef PROFILE

            profile_start = std::chrono::high_resolution_clock::now();
            #endif
            hipMemcpy(loss, d_loss, NP * NS * sizeof(double), hipMemcpyDeviceToHost);
            #ifdef PROFILE
            profile_stop = std::chrono::high_resolution_clock::now();
            profile_duration = profile_stop - profile_start; 
            std::cout << "inside cem_cuda:solve. hipMemcpy loss takes " << profile_duration.count() << "s" << std::endl; 
            std::cout << "inside cem_cuda:solve. 1000 steps of hipMemcpy loss takes " << 1000*profile_duration.count() << "s" << std::endl; 
            #endif
            for (unsigned int p = 0; p < NP; p++) {
                //std::cout<< "sorting... p=" << p <<std::endl;
                // copy loss to std::vector of std::pair. For sorting
                #ifdef PROFILE
                profile_start = std::chrono::high_resolution_clock::now();
                #endif
                for (unsigned int si = 0; si < NS; si++)
                {
                    loss_pair[si].first = loss[p*NS+si];
                    loss_pair[si].second = si;
                }

                sort(loss_pair.begin(), loss_pair.end());
                // copy sorted value from CPU to GPU
                for (unsigned int si = 0; si < NS; si++)
                {
                    loss[p*NS+si] = loss_pair[si].first;
                    loss_ind[p*NS+si] = loss_pair[si].second;
                }
                #ifdef PROFILE
                profile_stop = std::chrono::high_resolution_clock::now();
                profile_duration = profile_stop - profile_start; 
                std::cout << "inside cem_cuda:solve. sort takes " << profile_duration.count() << "s" << std::endl; 
                std::cout << "inside cem_cuda:solve. 1000 steps of sort takes " << 1000*profile_duration.count() << "s" << std::endl; 
                #endif


                // profile_start = std::chrono::high_resolution_clock::now();
                // thrust::sequence(loss_ind_ptr + NS * p, loss_ind_ptr + NS * p + NS);


                // thrust::sort_by_key(loss_ptr + NS * p, loss_ptr + NS * p + NS, loss_ind_ptr + NS * p);
                // profile_stop = std::chrono::high_resolution_clock::now();
                // profile_duration = profile_stop - profile_start; 
                // std::cout << "inside cem_cuda:solve. thrust calls takes " << profile_duration.count() << "s" << std::endl; 
                // std::cout << "inside cem_cuda:solve. 1000 steps of thrust calls takes " << 1000*profile_duration.count() << "s" << std::endl; 
        
            }
            #ifdef PROFILE
            profile_start = std::chrono::high_resolution_clock::now();
            #endif
            // copy sorted value from CPU to GPU
            hipMemcpy(d_loss_ind, loss_ind, NP*NS*sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(d_loss, loss, NP*NS*sizeof(double), hipMemcpyHostToDevice);
            #ifdef PROFILE
            profile_stop = std::chrono::high_resolution_clock::now();
            profile_duration = profile_stop - profile_start; 
            std::cout << "inside cem_cuda:solve. hipMemcpy loss to device takes " << profile_duration.count() << "s" << std::endl; 
            std::cout << "inside cem_cuda:solve. 1000 steps of hipMemcpy loss to device takes " << 1000*profile_duration.count() << "s" << std::endl; 
            #endif
            */



            //** Below uses approximate top-k method to bypass the memcpy overhead
            #ifdef PROFILE
            profile_start = std::chrono::high_resolution_clock::now();
            #endif

            dim3 grid_topk(1, 1, 1);    
            dim3 block_topk(NP, 1, N_ELITE);
            get_approx_topk_loss <<< grid_topk, block_topk >>>(d_loss, NS, d_top_k_loss, d_loss_ind, N_ELITE);
            #ifdef PROFILE
            profile_stop = std::chrono::high_resolution_clock::now();
            profile_duration = profile_stop - profile_start; 
            std::cout << "inside cem_cuda:solve. get_approx_topk_loss takes " << profile_duration.count() << "s" << std::endl; 
            std::cout << "inside cem_cuda:solve. 1000 steps of get_approx_topk_loss takes " << 1000*profile_duration.count() << "s" << std::endl; 
            #endif

            //** End of approximate top-k



            //std::cout<< "end of sorting." <<std::endl;
            #ifdef PROFILE

            profile_start = std::chrono::high_resolution_clock::now();
            #endif
            update_statistics<<<grid, block_pt >>>(d_control, d_time, d_mean_control, d_mean_time, d_std_control, d_std_time,
                d_loss_ind,  d_loss, NP, NS, NT, N_ELITE, d_best_ut);
            
            #ifdef PROFILE
            profile_stop = std::chrono::high_resolution_clock::now();
            profile_duration = profile_stop - profile_start; 
            std::cout << "inside cem_cuda:solve. update_statistics takes " << profile_duration.count() << "s" << std::endl; 
            std::cout << "inside cem_cuda:solve. 1000 steps of update_statistics takes " << 1000*profile_duration.count() << "s" << std::endl; 
            #endif

        }

        //std::cout << "copying from d_best_ut to best_u and best_t...\n" << std::endl;
        #ifdef PROFILE
        profile_start = std::chrono::high_resolution_clock::now();
        #endif
        hipMemcpy(best_u, d_best_ut, NP * NT * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(best_t, d_best_ut + NP * NT, NP * NT * sizeof(double), hipMemcpyDeviceToHost);
        #ifdef PROFILE
        profile_stop = std::chrono::high_resolution_clock::now();
        profile_duration = profile_stop - profile_start; 
        std::cout << "inside cem_cuda:solve. hipMemcpy of best_ut takes " << profile_duration.count() << "s" << std::endl; 
        std::cout << "inside cem_cuda:solve. 1000 steps of hipMemcpy of best_ut takes " << 1000*profile_duration.count() << "s" << std::endl; 
        #endif

        //std::cout << "inside CEM_CUDA::solve end." << std::endl;

    }

    
    unsigned int CEM_CUDA_acrobot::get_control_dimension(){
        return c_dim * NT;
    }

    unsigned int CEM_CUDA_acrobot::get_num_step(){
        return NT;
    }
}